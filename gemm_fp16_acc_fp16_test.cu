
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <chrono>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS Error: " << status << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(1); \
    } \
}

int main() {
    // 矩阵维度 (4090 适合大尺寸)
    int M = 2048, N = 2048, K = 2048;
    size_t bytes_A = M * K * sizeof(__half);
    size_t bytes_B = K * N * sizeof(__half);
    size_t bytes_C = M * N * sizeof(__half);

    // 分配设备内存
    __half *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, bytes_A));
    CHECK_CUDA(hipMalloc(&d_B, bytes_B));
    CHECK_CUDA(hipMalloc(&d_C, bytes_C));

    // 初始化 cuBLAS
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // 设置累加器为 FP16
    hipblasComputeType_t computeType = HIPBLAS_COMPUTE_16F;  // FP16 累加
    hipblasGemmAlgo_t algo = HIPBLAS_GEMM_DEFAULT;           // 自动选择算法

    // 标量参数 (FP16)
    __half alpha = __float2half(1.0f), beta = __float2half(0.0f);

    // 预热运行
    CHECK_CUBLAS(hipblasGemmEx(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        M, N, K,
        &alpha,
        d_A, HIP_R_16F, M,
        d_B, HIP_R_16F, K,
        &beta,
        d_C, HIP_R_16F, M,
        computeType, algo
    ));
    CHECK_CUDA(hipDeviceSynchronize());

    // 性能测试
    const int trials = 200;
    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < trials; ++i) {
        CHECK_CUBLAS(hipblasGemmEx(
            handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            M, N, K,
            &alpha,
            d_A, HIP_R_16F, M,
            d_B, HIP_R_16F, K,
            &beta,
            d_C, HIP_R_16F, M,
            computeType, algo
        ));
    }
    CHECK_CUDA(hipDeviceSynchronize());
    auto end = std::chrono::high_resolution_clock::now();

    // 计算性能
    float time_ms = std::chrono::duration<float>(end - start).count() * 1000 / trials;
    double flops = 2.0 * M * N * K * 1e-12; // TFLOPs
    double tflops = flops / (time_ms / 1000);

    std::cout << "Time: " << time_ms << " ms" << std::endl;
    std::cout << "Perf: " << tflops << " TFLOPS" << std::endl;

    // 清理
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    return 0;
}
